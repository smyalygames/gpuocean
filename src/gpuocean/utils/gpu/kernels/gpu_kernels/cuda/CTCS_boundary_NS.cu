#include "hip/hip_runtime.h"
/*
This software is part of GPU Ocean. 

Copyright (C) 2018-2019  SINTEF Digital
Copyright (C) 2018 Norwegian Meteorological Institute

These CUDA kernels implement boundary conditions for 
the Centered in Time, Centered in Space(leapfrog)
numerical scheme for the shallow water equations, described in 
L. P. Røed, "Documentation of simple ocean models for use in ensemble
predictions", Met no report 2012/3 and 2012/5.

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "common.cu"

// Boundary conditions are defined as
// 1: Closed wall
// 2: Periodic
// 3: Open (Flow Relaxation Scheme)
// 4: Open (Linear interpolation)

 // Fix north-south boundary before east-west (to get the corners right)
 extern "C" {
__global__ void boundaryEtaKernel_NS(
        // Discretization parameters
        int nx_, int ny_,
        int halo_x_, int halo_y_,
        int bc_north_, int bc_south_,
    
        // Data
        float* eta_ptr_, int eta_pitch_) {
    // Global thread sizes:
    // ti = {0, nx_+1}
    // tj = {0, 3},
    // thread 0 is index 0, thread 1 is index ny_+1, thread 2 and 3 idle
    
    // Index of cell within domain
    const int ti = blockIdx.x * blockDim.x + threadIdx.x;
    const int thread_id = blockIdx.y * blockDim.y + threadIdx.y;
    const int tj = (thread_id == 1) ? ny_+1 : thread_id;
    
    int opposite_row_index = ny_;
    //if (tj == ny_+1) {
    if ( (tj == ny_+1 && bc_north_ == 2) || (tj == 0 && bc_south_ == 1) ) {
        opposite_row_index = 1;
    }
    
    // Set ghost cells equal to inner neighbour's value
    if (((tj == 0     && bc_south_ < 3)  ||
         (tj == ny_+1 && bc_north_ < 3)) &&
        ti > -1 && ti < nx_+2) {
        float* ghost_row = (float*) ((char*) eta_ptr_ + eta_pitch_*tj);
        float* opposite_row = (float*) ((char*) eta_ptr_ + eta_pitch_*opposite_row_index);
        ghost_row[ti] = opposite_row[ti];
    }
}
} // extern "C"


// NS need to be called before EW!
extern "C" {
__global__ void boundaryUKernel_NS(
        // Discretization parameters
        int nx_, int ny_,
        int nx_halo_, int ny_halo_,
        int bc_north_, int bc_south_,

        // Data
        float* U_ptr_, int U_pitch_) {
    // Global thread sizes:
    // ti = {0, nx_+1}
    // tj = {0, 3},
    // thread 0 is index 0, thread 1 is index ny_+1, thread 2 and 3 idle
    
    // Index of cell within domain
    const int ti = blockIdx.x * blockDim.x + threadIdx.x;
    const int thread_id = blockIdx.y * blockDim.y + threadIdx.y;
    const int tj = (thread_id == 1) ? ny_+1 : thread_id;
    
    // Check if thread is in the domain:
    if (ti <= nx_+2 && tj <= ny_+1) {
        // The thread's row:
        float* u_row = (float*) ((char*) U_ptr_ + U_pitch_*tj);

        int opposite_row_index = ny_;
        if ( (tj == ny_+1 && bc_north_ == 2) || (tj == 0 && bc_south_ == 1) ) {
            opposite_row_index = 1;
        }
        
        if ( ((tj == 0     && bc_south_ < 3)  ||
              (tj == ny_+1 && bc_north_ < 3)) &&
              ti < nx_+3) {
              //ti > 0 && ti < nx_+1 ) {
            float* u_opposite_row = (float*) ((char*) U_ptr_ + U_pitch_*opposite_row_index);
            u_row[ti] = u_opposite_row[ti];
        }
    } 
}
} // extern "C"


// NS need to be called before EW!
extern "C" {
__global__ void boundaryVKernel_NS(
        // Discretization parameters
        int nx_, int ny_,
        int nx_halo_, int ny_halo_,
        int bc_north_, int bc_south_,

        // Data
        float* V_ptr_, int V_pitch_) {
    // Global thread sizes:
    // tj = {0, 3} 
    // thread 0 is index 0, thread 1 is index 1,
    // thread 2 is ny+1 and thread 3 is ny+2
    // ti = {0, nx_+1},
    
    // Index of cell within domain
    const int ti = blockIdx.x * blockDim.x + threadIdx.x;
    const int thread_id = blockIdx.y * blockDim.y + threadIdx.y;
    const int tj = (thread_id > 1) ? ny_ - 1 + thread_id : thread_id;


    // Check if thread is in the domain:
    if (ti <= nx_+1 && tj <= ny_+2) {   
        float* v_row = (float*) ((char*) V_ptr_ + V_pitch_*tj);

        
        if ( (tj < 2 && bc_south_ == 1 ) || (tj > ny_ && bc_north_ == 1) ) {
            if (tj == 0) {
                float* v_row_inner = (float*) ((char*) V_ptr_ + V_pitch_*2);
                v_row[ti] = -v_row_inner[ti];
            }
            else if (tj == ny_+2) {
                float* v_row_inner = (float*) ((char*) V_ptr_ + V_pitch_*ny_);
                v_row[ti] = -v_row_inner[ti];
            }
            else{
                v_row[ti] = 0.0f;
            }
        }
        else if (bc_north_ == 2) { // implicit bc_south_ == 2
            // Periodic
            int opposite_row_index = ny_;
            if (tj == ny_+2) {
                opposite_row_index = 2; // 2 and ny_+1 -> 1
            }
            if (tj == ny_+1) {
                opposite_row_index = 1;
            }
            
            if ( (tj == 0 || tj > ny_) && ti < nx_+2 ) {
                float* v_opposite_row = (float*) ((char*) V_ptr_ + V_pitch_*opposite_row_index);
                v_row[ti] = v_opposite_row[ti];
            }
        }
    }
}
} // extern "C"

